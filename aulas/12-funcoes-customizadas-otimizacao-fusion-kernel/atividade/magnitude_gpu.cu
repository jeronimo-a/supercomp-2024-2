#include "hip/hip_runtime.h"
#include <cmath>
#include <chrono>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

struct square {
    __host__ __device__
    float operator()(const float& x) const {
        return x * x;
    }
};

float magnitude(thrust::device_vector<float>& v) {
    float sum_of_squares = thrust::transform_reduce(v.begin(), v.end(), square(), 0.0f, thrust::plus<float>());
    return std::sqrt(sum_of_squares);
}

int main() {

    // lê o tamanho do arquivo de entrada
    int size;
    std::cin >> size;

    // lê o arquivo de entrada
    thrust::host_vector<float> v_host(size);
    for (int i = 0; i < size; i++) {
        std::cin >> v_host[i];
    }

    // copia os valores para a GPU
    thrust::device_vector<float> v(v_host);

    // inicia a marcação do tempo
    auto start = std::chrono::steady_clock::now();

    // realiza a operação
    float v_norm = magnitude(v);

    // termina a marcação do tempo
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> duration = end - start;

    // imprime os resultados
    std::cout << "Tamanho do vetor:  " << v_host.size() << std::endl;
    std::cout << "Norma do vetor:    " << v_norm << std::endl;
    std::cout << "Tempo de execução: " << duration.count() << " s" << std::endl;

    return 0;
}